#include "RTFEM/GPU/GPUMatrixMultiplication.cuh"

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <cstdlib>
#include <assert.h>
#include <stdexcept>

namespace rtfem {

template<class T>
GPUMatrixMultiplication<T>::GPUMatrixMultiplication() : d_A_(nullptr), n_(0) {}

template<class T>
GPUMatrixMultiplication<T>::~GPUMatrixMultiplication(){
    Terminate();
}

template<class T>
void GPUMatrixMultiplication<T>::PreSolve(T* A, int n){
    n_ = n;
    hipError_t cuda_error;
    hipblasStatus_t status;

    cuda_error = hipMalloc((void **) &d_A_, n_ * n_ * sizeof(*A));
    assert(hipSuccess == cuda_error);

    status = hipblasCreate(&cublas_handle_);
    assert(HIPBLAS_STATUS_SUCCESS == status);

    status = hipblasSetMatrix(n_, n_, sizeof(*A), A, n_, d_A_, n_);
    assert(HIPBLAS_STATUS_SUCCESS == status);
}

template<>
void GPUMatrixMultiplication<float>::PreSolve(float* A, int n){
    throw std::invalid_argument(
            "GPUMatrixMultiplication<float>::PreSolve not implemented");
}

template<class T>
void GPUMatrixMultiplication<T>::Solve(T* x, T alpha,
                                       T* y, T beta){
    T *d_x = nullptr;
    T *d_y = nullptr;

    hipError_t cuda_error;
    hipblasStatus_t status;

    cuda_error = hipMalloc((void **) &d_x, n_ * sizeof(*x));
    assert(hipSuccess == cuda_error);
    cuda_error = hipMalloc((void **) &d_y, n_ * sizeof(*y));
    assert(hipSuccess == cuda_error);

    status = hipblasSetVector(n_, sizeof(*x), x, 1, d_x, 1);
    assert(HIPBLAS_STATUS_SUCCESS == status);
    status = hipblasSetVector(n_, sizeof(*y), y, 1, d_y, 1);
    assert(HIPBLAS_STATUS_SUCCESS == status);

    status = hipblasDgemv(cublas_handle_, HIPBLAS_OP_N,
                         n_, n_,
                         &alpha, d_A_, n_, d_x, 1,
                         &beta, d_y, 1);
    assert(HIPBLAS_STATUS_SUCCESS == status);

    status = hipblasGetVector(n_, sizeof(*y), d_y, 1, y, 1);
    assert(HIPBLAS_STATUS_SUCCESS == status);

    hipFree(d_x);
    hipFree(d_y);
}

template<>
void GPUMatrixMultiplication<float>::Solve(float* x, float alpha,
                                           float* y, float beta){
    throw std::invalid_argument(
            "GPUMatrixMultiplication<float>::Solve not implemented");
}

template<class T>
void GPUMatrixMultiplication<T>::Terminate(){
    hipFree(d_A_);
    hipblasDestroy(cublas_handle_);
}

template
class GPUMatrixMultiplication<double>;
template
class GPUMatrixMultiplication<float>;

}
