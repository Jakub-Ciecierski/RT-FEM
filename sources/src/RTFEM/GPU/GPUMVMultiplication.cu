#include "RTFEM/GPU/GPUMVMultiplication.cuh"

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <cstdlib>
#include <assert.h>
#include <stdexcept>

namespace rtfem {

template<class T>
GPUMVMultiplication<T>::GPUMVMultiplication() : d_A_(nullptr), n_(0) {}

template<class T>
GPUMVMultiplication<T>::~GPUMVMultiplication(){
    Terminate();
}

template<class T>
void GPUMVMultiplication<T>::PreSolve(T* A, int n){
    n_ = n;
    hipError_t cuda_error;
    hipblasStatus_t status;

    cuda_error = hipMalloc((void **) &d_A_, n_ * n_ * sizeof(*A));
    assert(hipSuccess == cuda_error);

    status = hipblasCreate(&cublas_handle_);
    assert(HIPBLAS_STATUS_SUCCESS == status);

    status = hipblasSetMatrix(n_, n_, sizeof(*A), A, n_, d_A_, n_);
    assert(HIPBLAS_STATUS_SUCCESS == status);
}

template<>
void GPUMVMultiplication<float>::PreSolve(float* A, int n){
    throw std::invalid_argument(
            "GPUMVMultiplication<float>::PreSolve not implemented");
}

template<class T>
void GPUMVMultiplication<T>::Solve(T* x, T alpha,
                                       T* y, T beta){
    T *d_x = nullptr;
    T *d_y = nullptr;

    hipError_t cuda_error;
    hipblasStatus_t status;

    cuda_error = hipMalloc((void **) &d_x, n_ * sizeof(*x));
    assert(hipSuccess == cuda_error);
    cuda_error = hipMalloc((void **) &d_y, n_ * sizeof(*y));
    assert(hipSuccess == cuda_error);

    status = hipblasSetVector(n_, sizeof(*x), x, 1, d_x, 1);
    assert(HIPBLAS_STATUS_SUCCESS == status);
    status = hipblasSetVector(n_, sizeof(*y), y, 1, d_y, 1);
    assert(HIPBLAS_STATUS_SUCCESS == status);

    status = hipblasDgemv(cublas_handle_, HIPBLAS_OP_N,
                         n_, n_,
                         &alpha, d_A_, n_, d_x, 1,
                         &beta, d_y, 1);
    assert(HIPBLAS_STATUS_SUCCESS == status);

    status = hipblasGetVector(n_, sizeof(*y), d_y, 1, y, 1);
    assert(HIPBLAS_STATUS_SUCCESS == status);

    hipFree(d_x);
    hipFree(d_y);
}

template<>
void GPUMVMultiplication<float>::Solve(float *x, float alpha,
                                       float *y, float beta) {
    throw std::invalid_argument(
            "GPUMVMultiplication<float>::Solve not implemented");
}

template<class T>
void GPUMVMultiplication<T>::Terminate(){
    hipFree(d_A_);
    hipblasDestroy(cublas_handle_);
}

template
class GPUMVMultiplication<double>;
template
class GPUMVMultiplication<float>;

}
