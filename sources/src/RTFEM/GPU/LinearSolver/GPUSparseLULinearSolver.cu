#include "hip/hip_runtime.h"
#include "RTFEM/GPU/LinearSolver/GPUSparseLULinearSolver.cuh"

#include <RTFEM/DataStructure/SparseMatrixCSR.h>

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <math_functions.h>

namespace rtfem {

template<class T>
GPUSparseLULinearSolver<T>::GPUSparseLULinearSolver() :
        d_x(nullptr),
        d_z(nullptr),
        d_y(nullptr),
        pBuffer(nullptr),
        descr_L(nullptr),
        descr_U(nullptr),
        info_L(nullptr),
        info_U(nullptr){}

template<class T>
GPUSparseLULinearSolver<T>::~GPUSparseLULinearSolver(){}

template<class T>
void GPUSparseLULinearSolver<T>::PreSolve(const SparseMatrixCSR<T>& A){
    this->pre_solved_ = true;

    hipsparseStatus_t cusparseStatus;
    cusparseStatus = hipsparseCreate(&this->cusparseHandle);
    assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);

    /*******************************/
    /* FROM DENSE TO SPARSE MATRIX */
    /*******************************/
    // --- Descriptor for sparse matrix A
    cusparseStatus = hipsparseCreateMatDescr(&this->description);
    assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);
    cusparseStatus = hipsparseSetMatType(this->description,
                                        HIPSPARSE_MATRIX_TYPE_GENERAL);
    assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);
    cusparseStatus = hipsparseSetMatIndexBase(this->description,
                                             HIPSPARSE_INDEX_BASE_ZERO);
    assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);

    this->N = A.n();
    this->nnz = A.values().size();

    hipError_t cuda_error;
    // --- Device side sparse matrix
    cuda_error = hipMalloc(&this->d_val, this->nnz * sizeof(T));
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc(&this->d_row, (this->N + 1) * sizeof(int));
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc(&this->d_col, this->nnz * sizeof(int));
    assert(cuda_error == hipSuccess);

    cuda_error = hipMemcpy(this->d_col, A.columns_indices().data(),
               this->nnz*sizeof(int), hipMemcpyHostToDevice);
    assert(cuda_error == hipSuccess);
    cuda_error = hipMemcpy(this->d_row, A.row_extents().data(),
               (this->N+1)*sizeof(int), hipMemcpyHostToDevice);
    assert(cuda_error == hipSuccess);
    cuda_error = hipMemcpy(this->d_val, A.values().data(),
               this->nnz*sizeof(T), hipMemcpyHostToDevice);
    assert(cuda_error == hipSuccess);

    /******************************************/
    /* STEP 1: CREATE DESCRIPTORS FOR L AND U */
    /******************************************/
    hipsparseCreateMatDescr(&descr_L);
    hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr_L, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);
    hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_UNIT);

    hipsparseCreateMatDescr(&descr_U);
    hipsparseSetMatType(descr_U, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr_U, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatFillMode(descr_U, HIPSPARSE_FILL_MODE_UPPER);
    hipsparseSetMatDiagType(descr_U, HIPSPARSE_DIAG_TYPE_NON_UNIT);

    csrilu02Info_t info_A = 0;
    /**************************************************************************************************/
    /* STEP 2: QUERY HOW MUCH MEMORY USED IN LU FACTORIZATION AND THE TWO FOLLOWING SYSTEM INVERSIONS */
    /**************************************************************************************************/

    (hipsparseCreateCsrilu02Info(&info_A));
    (hipsparseCreateCsrsv2Info(&info_L));
    (hipsparseCreateCsrsv2Info(&info_U));

    int pBufferSize_M, pBufferSize_L, pBufferSize_U;
    cusparseStatus = (hipsparseDcsrilu02_bufferSize(this->cusparseHandle,
                                  this->N, this->nnz,
                                  this->description,
                                  this->d_val,
                                  this->d_row,
                                  this->d_col,
                                  info_A,
                                  &pBufferSize_M));
    assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);

    cusparseStatus = (hipsparseDcsrsv2_bufferSize(this->cusparseHandle,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                this->N,
                                this->nnz, descr_L,
                                this->d_val,
                                this->d_row,
                                this->d_col, info_L,
                                &pBufferSize_L));
    assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);

    cusparseStatus = (hipsparseDcsrsv2_bufferSize(this->cusparseHandle,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                this->N,
                                this->nnz, descr_U,
                                this->d_val,
                                this->d_row,
                                this->d_col,
                                info_U,
                                &pBufferSize_U));
    assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);

    int pBufferSize = max(pBufferSize_M, max(pBufferSize_L, pBufferSize_U));
    cuda_error = (hipMalloc((void**)&pBuffer, pBufferSize));
    assert(cuda_error == hipSuccess);

    /************************************************************************************************/
    /* STEP 3: ANALYZE THE THREE PROBLEMS: LU FACTORIZATION AND THE TWO FOLLOWING SYSTEM INVERSIONS */
    /************************************************************************************************/;
    cusparseStatus = (hipsparseDcsrilu02_analysis(this->cusparseHandle,
                                this->N,
                                this->nnz,
                                this->description,
                                this->d_val,
                                this->d_row,
                                this->d_col,
                                info_A, HIPSPARSE_SOLVE_POLICY_NO_LEVEL,
                                pBuffer));
    assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);

    int structural_zero;
    cusparseStatus = hipsparseXcsrilu02_zeroPivot(
            this->cusparseHandle, info_A, &structural_zero);
    //assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);

    cusparseStatus = (hipsparseDcsrsv2_analysis(this->cusparseHandle,
                              HIPSPARSE_OPERATION_NON_TRANSPOSE,
                              this->N, this->nnz, descr_L,
                              this->d_val,
                              this->d_row, this->d_col,
                              info_L,
                              HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer));
    assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);

    cusparseStatus = (hipsparseDcsrsv2_analysis(this->cusparseHandle,
                              HIPSPARSE_OPERATION_NON_TRANSPOSE,
                              this->N, this->nnz, descr_U,
                              this->d_val,
                              this->d_row,
                              this->d_col, info_U,
                              HIPSPARSE_SOLVE_POLICY_USE_LEVEL, pBuffer));
    assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);

    /************************************/
    /* STEP 4: FACTORIZATION: A = L * U */
    /************************************/

    int numerical_zero;
    cusparseStatus = hipsparseDcsrilu02(this->cusparseHandle,
                      this->N,
                      this->nnz,
                      this->description,
                      this->d_val,
                      this->d_row,
                      this->d_col, info_A,
                      HIPSPARSE_SOLVE_POLICY_NO_LEVEL,
                      pBuffer);
    assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);
    cusparseStatus = hipsparseXcsrilu02_zeroPivot(
            this->cusparseHandle, info_A, &numerical_zero);
    //assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);

    (hipMalloc(&d_z, this->N * sizeof(T)));
    (hipMalloc(&d_x, this->N * sizeof(T)));
    (hipMalloc(&d_y, this->N * sizeof(T)));
}

template<>
void GPUSparseLULinearSolver<float>::PreSolve(const SparseMatrixCSR<float>& A){
    throw std::invalid_argument(
            "GPUSparseLULinearSolver<float>::PreSolve not implemented");
}

template<class T>
void GPUSparseLULinearSolver<T>::Solve(const T* b, T* x){
    /*********************/
    /* STEP 5: L * z = x */
    /*********************/;

    (hipMemcpy(d_x, b, this->N * sizeof(T),
                hipMemcpyHostToDevice));

    hipsparseStatus_t cusparseStatus;

    const T alpha = 1.0;
    cusparseStatus = (hipsparseDcsrsv2_solve(this->cusparseHandle,
                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                           this->N,
                           this->nnz,
                           &alpha, descr_L,
                           this->d_val,
                           this->d_row,
                           this->d_col,
                           info_L,
                           d_x,
                           d_z,
                           HIPSPARSE_SOLVE_POLICY_NO_LEVEL,
                           pBuffer));
    assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);

    /*********************/
    /* STEP 5: U * y = z */
    /*********************/

    cusparseStatus = (hipsparseDcsrsv2_solve(
            this->cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            this->N,
            this->nnz,
            &alpha,
            descr_U,
            this->d_val,
            this->d_row,
            this->d_col,
            info_U,
            d_z,
            d_y,
            HIPSPARSE_SOLVE_POLICY_USE_LEVEL,
            pBuffer));
    assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);

    /********************************/
    /* MOVE THE RESULTS TO THE HOST */
    /********************************/
    hipMemcpy(x, d_y, this->N * sizeof(T), hipMemcpyDeviceToHost);
}

template<>
void GPUSparseLULinearSolver<float>::Solve(const float* b, float* x){
    throw std::invalid_argument(
            "GPUSparseLULinearSolver<float>::Solve not implemente");
}

template<class T>
void GPUSparseLULinearSolver<T>::Terminate(){
    if(this->pre_solved_){
        if(descr_L)
            hipsparseDestroyMatDescr(descr_L);
        if(descr_U)
            hipsparseDestroyMatDescr(descr_U);
        if(this->description)
            hipsparseDestroyMatDescr(this->description);

        if(this->cusparseHandle)
            hipsparseDestroy(this->cusparseHandle);
        if(this->d_col)
            hipFree(this->d_col);
        if(this->d_row)
            hipFree(this->d_row);
        if(this->d_val)
            hipFree(this->d_val);

        if(d_z)
            hipFree(d_z);
        if(d_x)
            hipFree(d_x);
        if(d_y)
            hipFree(d_y);
    }
}

template
class GPUSparseLULinearSolver<float>;
template
class GPUSparseLULinearSolver<double>;

}