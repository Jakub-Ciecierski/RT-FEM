#include "RTFEM/GPU/LinearSolver/GPUSparseCGPrecondLinearSolver.cuh"

#include <RTFEM/DataStructure/SparseMatrixCSR.h>

#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <assert.h>
#include <stdexcept>
#include <iostream>

namespace rtfem {

template<class T>
GPUSparseCGPrecondLinearSolver<T>::GPUSparseCGPrecondLinearSolver() :
        d_y(nullptr),
        d_omega(nullptr),
        d_valsILU0(nullptr),
        d_zm1(nullptr),
        d_zm2(nullptr),
        d_rm2(nullptr),
        infoA(nullptr),
        info_u(nullptr),
        descrL(nullptr),
        descrU(nullptr) {}

template<class T>
GPUSparseCGPrecondLinearSolver<T>::~GPUSparseCGPrecondLinearSolver(){
    Terminate();
}

template<class T>
void GPUSparseCGPrecondLinearSolver<T>::PreSolve(const SparseMatrixCSR<T>& A){
    this->pre_solved_ = true;

    this->N = A.n();
    this->nnz = A.values().size();
    
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&this->cublasHandle);
    assert(hipblasStatus_t == HIPBLAS_STATUS_SUCCESS);

    /* Create CUSPARSE context */
    hipsparseStatus_t cusparseStatus;
    cusparseStatus = hipsparseCreate(&this->cusparseHandle);
    assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);

    /* Description of the A matrix*/
    cusparseStatus = hipsparseCreateMatDescr(&this->description);
    assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);

    /* Define the properties of the matrix */
    hipsparseSetMatType(this->description, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(this->description, HIPSPARSE_INDEX_BASE_ZERO);

    /* Allocate required memory */
    hipMalloc((void **)&this->d_col, this->nnz*sizeof(int));
    hipMalloc((void **)&this->d_row, (this->N+1)*sizeof(int));
    hipMalloc((void **)&this->d_val, this->nnz*sizeof(T));
    hipMalloc((void **)&this->d_x, this->N*sizeof(T));
    hipMalloc((void **)&this->d_y, this->N*sizeof(T));
    hipMalloc((void **)&this->d_r, this->N*sizeof(T));
    hipMalloc((void **)&this->d_p, this->N*sizeof(T));
    hipMalloc((void **)&this->d_omega, this->N*sizeof(T));
    hipMalloc((void **)&this->d_valsILU0, this->nnz*sizeof(T));
    hipMalloc((void **)&this->d_zm1, (this->N)*sizeof(T));
    hipMalloc((void **)&this->d_zm2, (this->N)*sizeof(T));
    hipMalloc((void **)&this->d_rm2, (this->N)*sizeof(T));
    
    hipMemcpy(this->d_col, A.columns_indices().data(), this->nnz * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(this->d_row, A.row_extents().data(), (this->N + 1) * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(this->d_val, A.values().data(), this->nnz * sizeof(T),
               hipMemcpyHostToDevice);
    
    /* create the analysis info object for t`he A matrix */
    cusparseStatus = cusparseCreateSolveAnalysisInfo(&infoA);
    assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);

    /* Perform the analysis for the Non-Transpose case */
    cusparseStatus = cusparseDcsrsv_analysis(this->cusparseHandle,
                                             HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                             this->N,
                                             this->nnz,
                                             this->description,
                                             this->d_val,
                                             this->d_row,
                                             this->d_col,
                                             infoA);
    assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);

    /* Copy A data to ILU0 vals as input*/
    hipMemcpy(d_valsILU0,
               this->d_val,
               this->nnz*sizeof(T), hipMemcpyDeviceToDevice);

    /* generate the Incomplete LU factor H for the matrix A using cudsparseScsrilu0 */
    cusparseStatus = cusparseDcsrilu0(this->cusparseHandle,
                                      HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                      this->N,
                                      this->description,
                                      d_valsILU0,
                                      this->d_row,
                                      this->d_col,
                                      infoA);
    assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);

    /* Create info objects for the ILU0 preconditioner */
    cusparseCreateSolveAnalysisInfo(&info_u);

    cusparseStatus = hipsparseCreateMatDescr(&descrL);
    assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);
    hipsparseSetMatType(descrL, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrL, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatFillMode(descrL, HIPSPARSE_FILL_MODE_LOWER);
    hipsparseSetMatDiagType(descrL, HIPSPARSE_DIAG_TYPE_UNIT);

    cusparseStatus = hipsparseCreateMatDescr(&descrU);
    assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);

    hipsparseSetMatType(descrU,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrU,HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatFillMode(descrU, HIPSPARSE_FILL_MODE_UPPER);
    hipsparseSetMatDiagType(descrU, HIPSPARSE_DIAG_TYPE_NON_UNIT);
    cusparseStatus = cusparseDcsrsv_analysis(this->cusparseHandle,
                                             HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                             this->N,
                                             this->nnz,
                                             descrU,
                                             this->d_val,
                                             this->d_row,
                                             this->d_col,
                                             info_u);
    assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);
}

template<>
void GPUSparseCGPrecondLinearSolver<float>::PreSolve(
        const SparseMatrixCSR<float>& A){
    throw std::invalid_argument(
            "GPUSparsePreCondLinearSolver<float>::PreSolve not implemented");
}

template<class T>
void GPUSparseCGPrecondLinearSolver<T>::Solve(const T* b, T* x){
    const int max_iter = 1000;
    const T tol = 1e-5f;

    int k = 0;
    T r1, alpha, beta;
    T numerator, denominator, nalpha;
    const T floatone = 1.0;
    const T floatzero = 0.0;

    hipMemcpy(this->d_x, x, this->N*sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(this->d_r, b, this->N*sizeof(T), hipMemcpyHostToDevice);

    int nzILU0 = 2*this->N-1;

    for (int i = 0; i < this->N; i++)
    {
        x[i] = 0.0;
    }

    hipMemcpy(this->d_r, b, this->N*sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(this->d_x, x, this->N*sizeof(T), hipMemcpyHostToDevice);

    k = 0;
    hipblasDdot(this->cublasHandle,
               this->N,
               this->d_r, 1,
               this->d_r, 1, &r1);
    hipsparseStatus_t cusparseStatus;
    while (r1 > tol*tol && k <= max_iter)
    {
        // Forward Solve, we can re-use infoA since the sparsity pattern of A matches that of L
        cusparseStatus = cusparseDcsrsv_solve(this->cusparseHandle,
                                              HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                              this->N, &floatone, descrL,
                                              d_valsILU0,
                                              this->d_row,
                                              this->d_col,
                                              this->infoA,
                                              this->d_r, d_y);
        assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);

        // Back Substitution
        cusparseStatus = cusparseDcsrsv_solve(this->cusparseHandle,
                                              HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                              this->N, &floatone, descrU,
                                              d_valsILU0,
                                              this->d_row,
                                              this->d_col,
                                              this->info_u,
                                              this->d_y, d_zm1);
        assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);

        k++;

        if (k == 1)
        {
            hipblasDcopy(this->cublasHandle, this->N, d_zm1, 1,
                        this->d_p, 1);
        }
        else
        {
            hipblasDdot(this->cublasHandle,
                       this->N,
                       this->d_r, 1, d_zm1, 1, &numerator);
            hipblasDdot(this->cublasHandle,
                       this->N, d_rm2, 1, d_zm2, 1, &denominator);
            beta = numerator/denominator;
            hipblasDscal(this->cublasHandle, this->N, &beta, this->d_p, 1);
            hipblasDaxpy(this->cublasHandle, this->N,
                        &floatone, d_zm1, 1, this->d_p, 1) ;
        }

        hipsparseDcsrmv(this->cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                       this->N, this->N,
                       nzILU0, &floatone, descrU,
                       this->d_val, this->d_row, this->d_col, this->d_p,
                       &floatzero, d_omega);
        hipblasDdot(this->cublasHandle,
                   this->N, this->d_r, 1, d_zm1, 1, &numerator);
        hipblasDdot(this->cublasHandle,
                   this->N, this->d_p, 1, d_omega, 1, &denominator);
        alpha = numerator / denominator;
        hipblasDaxpy(this->cublasHandle,
                    this->N, &alpha,
                    this->d_p, 1, this->d_x, 1);
        hipblasDcopy(this->cublasHandle,
                    this->N, this->d_r, 1, d_rm2, 1);
        hipblasDcopy(this->cublasHandle,
                    this->N, d_zm1, 1, d_zm2, 1);
        nalpha = -alpha;
        hipblasDaxpy(this->cublasHandle, this->N, &nalpha,
                    d_omega, 1, this->d_r, 1);
        hipblasDdot(this->cublasHandle, this->N, this->d_r, 1,
                   this->d_r, 1, &r1);
    }

    hipMemcpy(x, this->d_x, this->N*sizeof(T), hipMemcpyDeviceToHost);
}

template<>
void GPUSparseCGPrecondLinearSolver<float>::Solve(
        const float* B, float* x){
    throw std::invalid_argument(
            "GPUSparsePreCondLinearSolver<float>::Solve not implemented");
}

template<class T>
void GPUSparseCGPrecondLinearSolver<T>::Terminate(){
    if(this->pre_solved_) {
        /* Destroy parameters */
        cusparseDestroySolveAnalysisInfo(infoA);
        cusparseDestroySolveAnalysisInfo(info_u);

        hipsparseDestroyMatDescr(descrL);
        hipsparseDestroyMatDescr(descrU);

        if(this->d_y)
            hipFree(this->d_y);
        if(this->d_omega)
            hipFree(this->d_omega);
        if(this->d_valsILU0)
            hipFree(this->d_valsILU0);
        if(this->d_zm1)
            hipFree(this->d_zm1);
        if(this->d_zm2)
            hipFree(this->d_zm2);
        if(this->d_rm2)
            hipFree(this->d_rm2);
    }
}

template
class GPUSparseCGPrecondLinearSolver<double>;
template
class GPUSparseCGPrecondLinearSolver<float>;

}