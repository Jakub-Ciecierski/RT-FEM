#include "RTFEM/GPU/LinearSolver/GPUSparseCGLinearSolver.cuh"

#include <RTFEM/DataStructure/SparseMatrixCSR.h>

#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <assert.h>
#include <stdexcept>

namespace rtfem {

template<class T>
GPUSparseCGLinearSolver<T>::GPUSparseCGLinearSolver() :
        d_x(nullptr),
        d_r(nullptr),
        d_p(nullptr),
        d_Ax(nullptr){}

template<class T>
GPUSparseCGLinearSolver<T>::~GPUSparseCGLinearSolver(){
    Terminate();
}

template<class T>
void GPUSparseCGLinearSolver<T>::PreSolve(const SparseMatrixCSR<T>& A){
    this->pre_solved_ = true;
    this->N = A.n();
    this->nnz = A.values().size();

    hipblasStatus_t hipblasStatus_t;
    hipError_t cuda_error;
    hipblasStatus_t = hipblasCreate(&this->cublasHandle);
    assert(hipblasStatus_t == HIPBLAS_STATUS_SUCCESS);

    hipsparseStatus_t cusparseStatus;
    cusparseStatus = hipsparseCreate(&this->cusparseHandle);
    assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);

    cusparseStatus = hipsparseCreateMatDescr(&this->description);
    assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);

    hipsparseSetMatType(this->description, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(this->description, HIPSPARSE_INDEX_BASE_ZERO);

    cuda_error = hipMalloc((void **)&this->d_col, this->nnz*sizeof(int));
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc((void **)&this->d_row, (this->N+1)*sizeof(int));
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc((void **)&this->d_val, this->nnz*sizeof(T));
    assert(cuda_error == hipSuccess);

    cuda_error = hipMalloc((void **)&d_x, this->N*sizeof(T));
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc((void **)&d_r, this->N*sizeof(T));
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc((void **)&d_p, this->N*sizeof(T));
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc((void **)&d_Ax, this->N*sizeof(T));
    assert(cuda_error == hipSuccess);

    hipMemcpy(this->d_col, A.columns_indices().data(),
               this->nnz*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(this->d_row, A.row_extents().data(),
               (this->N+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(this->d_val, A.values().data(),
               this->nnz*sizeof(T), hipMemcpyHostToDevice);
}

template<>
void GPUSparseCGLinearSolver<float>::PreSolve(const SparseMatrixCSR<float>& A){
    throw std::invalid_argument(
            "GPUSparseLinearSolver<float>::PreSolve not implemented");
}

template<class T>
void GPUSparseCGLinearSolver<T>::Solve(
        const T* B, T* x){
    const T tol = 1e-5f;
    const int max_iter = 1000;
    T a, b, na, r0, r1;
    T dot;
    int k;
    T alpha, beta, alpham1;

    hipMemcpy(d_x, x, this->N*sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_r, B, this->N*sizeof(T), hipMemcpyHostToDevice);

    alpha = 1.0;
    alpham1 = -1.0;
    beta = 0.0;
    r0 = 0.0;

    hipsparseDcsrmv(this->cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE,
                   this->N, this->N, this->nnz,
                   &alpha, this->description,
                   this->d_val, this->d_row, this->d_col, d_x, &beta, d_Ax);

    hipblasDaxpy(this->cublasHandle, this->N,
                &alpham1, d_Ax, 1, d_r, 1);
    hipblasDdot(this->cublasHandle,
               this->N, d_r, 1, d_r, 1, &r1);

    k = 1;

    while (r1 > tol*tol && k <= max_iter)
    {
        if (k > 1)
        {
            b = r1 / r0;
            hipblasDscal(this->cublasHandle, this->N, &b, d_p, 1);
            hipblasDaxpy(this->cublasHandle, this->N, &alpha, d_r, 1, d_p, 1);
        }
        else
        {
            hipblasDcopy(this->cublasHandle, this->N, d_r, 1, d_p, 1);
        }

        hipsparseDcsrmv(this->cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                       this->N, this->N, this->nnz,
                       &alpha, this->description, this->d_val,
                       this->d_row, this->d_col, d_p,
                       &beta, d_Ax);
        hipblasDdot(this->cublasHandle, this->N, d_p, 1, d_Ax, 1, &dot);
        a = r1 / dot;

        hipblasDaxpy(this->cublasHandle, this->N, &a, d_p, 1, d_x, 1);
        na = -a;
        hipblasDaxpy(this->cublasHandle, this->N, &na, d_Ax, 1, d_r, 1);

        r0 = r1;
        hipblasDdot(this->cublasHandle, this->N, d_r, 1, d_r, 1, &r1);
        hipDeviceSynchronize();
        k++;
    }

    hipMemcpy(x, d_x, this->N*sizeof(T), hipMemcpyDeviceToHost);
}

template<>
void GPUSparseCGLinearSolver<float>::Solve(
        const float* b, float* x){
    throw std::invalid_argument(
            "GPUSparseLinearSolver<float>::Solve not implemented");
}

template<class T>
void GPUSparseCGLinearSolver<T>::Terminate(){
    if(this->pre_solved_) {
        if(this->description)
            hipsparseDestroyMatDescr(this->description);

        if(this->cusparseHandle)
            hipsparseDestroy(this->cusparseHandle);
        if(this->cublasHandle)
            hipblasDestroy(this->cublasHandle);
        if(this->d_col)
            hipFree(this->d_col);
        if(this->d_row)
            hipFree(this->d_row);
        if(this->d_val)
            hipFree(this->d_val);
        if(d_x)
            hipFree(d_x);
        if(d_r)
            hipFree(d_r);
        if(d_p)
            hipFree(d_p);
        if(d_Ax)
            hipFree(d_Ax);
    }
}

template
class GPUSparseCGLinearSolver<float>;
template
class GPUSparseCGLinearSolver<double>;

}