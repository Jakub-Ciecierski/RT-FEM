#include "RTFEM/GPU/LinearSolver/GPUSparseLinearSolver.cuh"

#include <RTFEM/DataStructure/SparseMatrixCSR.h>

#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <assert.h>
#include <stdexcept>

namespace rtfem {

template<class T>
GPUSparseLinearSolver<T>::GPUSparseLinearSolver() :
        d_col(nullptr),
        d_row(nullptr),
        d_val(nullptr),
        N(0),
        nnz(0),
        d_x(nullptr),
        d_r(nullptr),
        d_p(nullptr),
        d_Ax(nullptr),
        cusparseHandle(nullptr),
        cublasHandle(nullptr),
        description(nullptr),
        pre_solved_(false){}

template<class T>
GPUSparseLinearSolver<T>::~GPUSparseLinearSolver(){
    Terminate();
}

template<class T>
void GPUSparseLinearSolver<T>::PreSolve(const SparseMatrixCSR<T>& A){
    pre_solved_ = true;
    N = A.n();
    nnz = A.values().size();

    hipblasStatus_t hipblasStatus_t;
    hipError_t cuda_error;
    hipblasStatus_t = hipblasCreate(&cublasHandle);
    assert(hipblasStatus_t == HIPBLAS_STATUS_SUCCESS);

    hipsparseStatus_t cusparseStatus;
    cusparseStatus = hipsparseCreate(&cusparseHandle);
    assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);

    cusparseStatus = hipsparseCreateMatDescr(&description);
    assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);

    hipsparseSetMatType(description, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(description, HIPSPARSE_INDEX_BASE_ZERO);

    cuda_error = hipMalloc((void **)&d_col, nnz*sizeof(int));
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc((void **)&d_row, (N+1)*sizeof(int));
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc((void **)&d_val, nnz*sizeof(double));
    assert(cuda_error == hipSuccess);

    cuda_error = hipMalloc((void **)&d_x, N*sizeof(double));
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc((void **)&d_r, N*sizeof(double));
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc((void **)&d_p, N*sizeof(double));
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc((void **)&d_Ax, N*sizeof(double));
    assert(cuda_error == hipSuccess);

    hipMemcpy(d_col, A.columns_indices().data(),
               nnz*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_row, A.row_extents().data(),
               (N+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_val, A.values().data(),
               nnz*sizeof(double), hipMemcpyHostToDevice);
}

template<>
void GPUSparseLinearSolver<float>::PreSolve(const SparseMatrixCSR<float>& A){
    throw std::invalid_argument(
            "GPUSparseLinearSolver<float>::PreSolve not implemented");
}

template<class T>
void GPUSparseLinearSolver<T>::Solve(
        const T* B, T* x){
    const T tol = 1e-5f;
    const int max_iter = 1000;
    T a, b, na, r0, r1;
    T dot;
    int k;
    T alpha, beta, alpham1;

    hipMemcpy(d_x, x, N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_r, B, N*sizeof(double), hipMemcpyHostToDevice);

    alpha = 1.0;
    alpham1 = -1.0;
    beta = 0.0;
    r0 = 0.0;

    hipsparseDcsrmv(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE,
                   N, N, nnz,
                   &alpha, description, d_val, d_row, d_col, d_x, &beta, d_Ax);

    hipblasDaxpy(cublasHandle, N, &alpham1, d_Ax, 1, d_r, 1);
    hipblasDdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);

    k = 1;

    while (r1 > tol*tol && k <= max_iter)
    {
        if (k > 1)
        {
            b = r1 / r0;
            hipblasDscal(cublasHandle, N, &b, d_p, 1);
            hipblasDaxpy(cublasHandle, N, &alpha, d_r, 1, d_p, 1);
        }
        else
        {
            hipblasDcopy(cublasHandle, N, d_r, 1, d_p, 1);
        }

        hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N,
                       N, nnz, &alpha, description, d_val, d_row, d_col, d_p,
                       &beta, d_Ax);
        hipblasDdot(cublasHandle, N, d_p, 1, d_Ax, 1, &dot);
        a = r1 / dot;

        hipblasDaxpy(cublasHandle, N, &a, d_p, 1, d_x, 1);
        na = -a;
        hipblasDaxpy(cublasHandle, N, &na, d_Ax, 1, d_r, 1);

        r0 = r1;
        hipblasDdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);
        hipDeviceSynchronize();
        k++;
    }

    hipMemcpy(x, d_x, N*sizeof(double), hipMemcpyDeviceToHost);
}

template<>
void GPUSparseLinearSolver<float>::Solve(
        const float* b, float* x){
    throw std::invalid_argument(
            "GPUSparseLinearSolver<float>::Solve not implemented");
}

template<class T>
void GPUSparseLinearSolver<T>::Terminate(){
    if(pre_solved_) {
        if(cusparseHandle)
            hipsparseDestroy(cusparseHandle);
        if(cublasHandle)
            hipblasDestroy(cublasHandle);
        if(d_col)
            hipFree(d_col);
        if(d_row)
            hipFree(d_row);
        if(d_val)
            hipFree(d_val);
        if(d_x)
            hipFree(d_x);
        if(d_r)
            hipFree(d_r);
        if(d_p)
            hipFree(d_p);
        if(d_Ax)
            hipFree(d_Ax);
    }
}

template
class GPUSparseLinearSolver<float>;
template
class GPUSparseLinearSolver<double>;

}