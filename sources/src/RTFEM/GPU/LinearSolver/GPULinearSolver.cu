#include "RTFEM/GPU/LinearSolver/GPULinearSolver.cuh"

#include <hipsolver.h>

#include <assert.h>
#include <cstdlib>
#include <cstdio>
#include <stdexcept>

namespace rtfem {

template <class T>
CUDA_HOST_MEMBER
void GPULinearSolver<T>::Solve(T* A, T* b, int n, T* x){
    hipsolverHandle_t cusolverH = nullptr;
    hipStream_t stream = nullptr;

    // Host
    T* LU = (T*)malloc(sizeof(T) * n * n);
    int* Ipivot = (int*)malloc(sizeof(int) * n);
    int info = 0;

    // Device
    T* d_A = nullptr;
    T* d_b = nullptr;
    int* d_pivot = nullptr;
    int* d_info = nullptr;
    int lwork = 0;
    T* d_work = nullptr;
    hipError_t cuda_error = hipSuccess;

    /* step 1: create cusolver handle, bind a stream */
    auto status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    cuda_error = hipStreamCreateWithFlags(&stream,
                                           hipStreamNonBlocking);
    assert(hipSuccess == cuda_error);

    status = hipsolverSetStream(cusolverH, stream);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    /* step 2: copy A to device */
    cuda_error = hipMalloc ((void**)&d_A, sizeof(T) * n * n);
    assert(hipSuccess == cuda_error);
    cuda_error = hipMalloc ((void**)&d_b, sizeof(T) * n);
    assert(hipSuccess == cuda_error);
    cuda_error = hipMalloc ((void**)&d_pivot, sizeof(int) * n);
    assert(hipSuccess == cuda_error);
    cuda_error = hipMalloc ((void**)&d_info, sizeof(int));
    assert(hipSuccess == cuda_error);

    cuda_error = hipMemcpy(d_A, A, sizeof(T)*n*n, hipMemcpyHostToDevice);
    assert(hipSuccess == cuda_error);
    cuda_error = hipMemcpy(d_b, b, sizeof(T)*n, hipMemcpyHostToDevice);
    assert(hipSuccess == cuda_error);

    /* step 3: query working space of getrf */
    status = hipsolverDnDgetrf_bufferSize(
        cusolverH,
        n,
        n,
        d_A,
        n,
        &lwork);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    cuda_error = hipMalloc((void**)&d_work, sizeof(T)*lwork);
    assert(hipSuccess == cuda_error);

    /* step 4: LU factorization */
    status = hipsolverDnDgetrf(
        cusolverH,
        n,
        n,
        d_A,
        n,
        d_work,
        d_pivot,
        d_info);
    cuda_error = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cuda_error);

    cuda_error = hipMemcpy(Ipivot , d_pivot, sizeof(int)*n,
                            hipMemcpyDeviceToHost);
    assert(hipSuccess == cuda_error);

    cuda_error = hipMemcpy(LU, d_A, sizeof(T)*n*n,
                           hipMemcpyDeviceToHost);
    assert(hipSuccess == cuda_error);

    cuda_error = hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cuda_error);

    if ( 0 > info ){
        printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }


    /*
     * step 5: solve A*X = B
     *       | 1 |       | -0.3333 |
     *   B = | 2 |,  X = |  0.6667 |
     *       | 3 |       |  0      |
     *
     */
    status = hipsolverDnDgetrs(
        cusolverH,
        HIPBLAS_OP_N,
        n,
        1, /* nrhs */
        d_A,
        n,
        d_pivot,
        d_b,
        n,
        d_info);

    cuda_error = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cuda_error);

    cuda_error = hipMemcpy(x, d_b, sizeof(T)*n,
                            hipMemcpyDeviceToHost);
    assert(hipSuccess == cuda_error);

    /* free resources */
    if (d_A    ) hipFree(d_A);
    if (d_b    ) hipFree(d_b);
    if (d_pivot) hipFree(d_pivot);
    if (d_info ) hipFree(d_info);
    if (d_work ) hipFree(d_work);

    if(LU)
        free(LU);
    if(Ipivot)
        free(Ipivot);

    if (cusolverH   ) hipsolverDnDestroy(cusolverH);
    if (stream      ) hipStreamDestroy(stream);

    //hipDeviceReset();
}

template <>
CUDA_HOST_MEMBER
void GPULinearSolver<float>::Solve(float* A, float* b, int n, float* x){
    throw std::invalid_argument(
            "GPULinearSolver<float>::Solve not implemented");
}

template
class GPULinearSolver<double>;
template
class GPULinearSolver<float>;

}