#include "RTFEM/GPU/LinearSolver/GPULinearSolver.cuh"

#include <assert.h>
#include <cstdlib>
#include <cstdio>
#include <stdexcept>

#include <hipsolver.h>

namespace rtfem {

template<class T>
GPULinearSolver<T>::GPULinearSolver() :
    d_A(nullptr),
    d_b(nullptr),
    d_pivot(nullptr),
    d_info(nullptr),
    d_work(nullptr),
    cusolverH(nullptr),
    stream(nullptr) {}

template <class T>
GPULinearSolver<T>::~GPULinearSolver(){
    Terminate();
}

template <class T>
CUDA_HOST_MEMBER
void GPULinearSolver<T>::PreSolve(T* A, int n){
    n_ = n;

    // Host
    int info = 0;
    int lwork = 0;

    hipError_t cuda_error = hipSuccess;

    /* step 1: create cusolver handle, bind a stream */
    auto status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    cuda_error = hipStreamCreateWithFlags(&stream,
                                           hipStreamNonBlocking);
    assert(hipSuccess == cuda_error);

    status = hipsolverSetStream(cusolverH, stream);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    /* step 2: copy A to device */
    cuda_error = hipMalloc ((void**)&d_A, sizeof(T) * n * n);
    assert(hipSuccess == cuda_error);
    cuda_error = hipMalloc ((void**)&d_b, sizeof(T) * n);
    assert(hipSuccess == cuda_error);
    cuda_error = hipMalloc ((void**)&d_pivot, sizeof(int) * n);
    assert(hipSuccess == cuda_error);
    cuda_error = hipMalloc ((void**)&d_info, sizeof(int));
    assert(hipSuccess == cuda_error);

    cuda_error = hipMemcpy(d_A, A, sizeof(T)*n*n, hipMemcpyHostToDevice);
    assert(hipSuccess == cuda_error);

    /* step 3: query working space of getrf */
    status = hipsolverDnDgetrf_bufferSize(
        cusolverH,
        n,
        n,
        d_A,
        n,
        &lwork);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    cuda_error = hipMalloc((void**)&d_work, sizeof(T)*lwork);
    assert(hipSuccess == cuda_error);

    /* step 4: LU factorization */
    status = hipsolverDnDgetrf(
        cusolverH,
        n,
        n,
        d_A,
        n,
        d_work,
        d_pivot,
        d_info);
    cuda_error = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cuda_error);

    cuda_error = hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cuda_error);

    if ( 0 > info ){
        printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }

}

template <>
CUDA_HOST_MEMBER
void GPULinearSolver<float>::PreSolve(float* A, int n){
    throw std::invalid_argument(
        "GPULinearSolver<float>::PreSolve not implemented");
}

template <class T>
CUDA_HOST_MEMBER
void GPULinearSolver<T>::Solve(const T* b, int n, T* x){
    hipError_t cuda_error = hipSuccess;

    cuda_error = hipMemcpy(d_b, b, sizeof(T)*n, hipMemcpyHostToDevice);
    assert(hipSuccess == cuda_error);

    auto status = hipsolverDnDgetrs(
        cusolverH,
        HIPBLAS_OP_N,
        n,
        1, /* nrhs */
        d_A,
        n,
        d_pivot,
        d_b,
        n,
        d_info);

    cuda_error = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cuda_error);

    cuda_error = hipMemcpy(x, d_b, sizeof(T)*n,
                            hipMemcpyDeviceToHost);
    assert(hipSuccess == cuda_error);
}

template <>
CUDA_HOST_MEMBER
void GPULinearSolver<float>::Solve(const float* b, int n, float* x){
    throw std::invalid_argument(
            "GPULinearSolver<float>::Solve not implemented");
}

template <class T>
CUDA_HOST_MEMBER
void GPULinearSolver<T>::Terminate(){
    if (d_A    ) hipFree(d_A);
    if (d_b    ) hipFree(d_b);
    if (d_pivot) hipFree(d_pivot);
    if (d_info ) hipFree(d_info);
    if (d_work ) hipFree(d_work);

    if (cusolverH   ) hipsolverDnDestroy(cusolverH);
    if (stream      ) hipStreamDestroy(stream);
}

template
class GPULinearSolver<double>;
template
class GPULinearSolver<float>;

}