#include "RTFEM/GPU/LinearSolver/GPULinearSolver.cuh"

#include <hipsolver.h>

#include <assert.h>
#include <cstdlib>
#include <cstdio>

namespace rtfem {

CUDA_HOST_MEMBER
void GPULinearSolver::Solve(double* A, double* b, int n, double* x){
    hipsolverHandle_t cusolverH = nullptr;
    hipStream_t stream = nullptr;

    // Host
    double* LU = (double*)malloc(sizeof(double) * n * n);
    int* Ipivot = (int*)malloc(sizeof(int) * n);
    int info = 0;

    // Device
    double* d_A = nullptr;
    double* d_b = nullptr;
    int* d_pivot = nullptr;
    int* d_info = nullptr;
    int lwork = 0;
    double* d_work = nullptr;
    hipError_t cuda_error = hipSuccess;

    /* step 1: create cusolver handle, bind a stream */
    auto status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    cuda_error = hipStreamCreateWithFlags(&stream,
                                           hipStreamNonBlocking);
    assert(hipSuccess == cuda_error);

    status = hipsolverSetStream(cusolverH, stream);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    /* step 2: copy A to device */
    cuda_error = hipMalloc ((void**)&d_A, sizeof(double) * n * n);
    assert(hipSuccess == cuda_error);
    cuda_error = hipMalloc ((void**)&d_b, sizeof(double) * n);
    assert(hipSuccess == cuda_error);
    cuda_error = hipMalloc ((void**)&d_pivot, sizeof(int) * n);
    assert(hipSuccess == cuda_error);
    cuda_error = hipMalloc ((void**)&d_info, sizeof(int));
    assert(hipSuccess == cuda_error);

    cuda_error = hipMemcpy(d_A, A, sizeof(double)*n*n, hipMemcpyHostToDevice);
    assert(hipSuccess == cuda_error);
    cuda_error = hipMemcpy(d_b, b, sizeof(double)*n, hipMemcpyHostToDevice);
    assert(hipSuccess == cuda_error);

    /* step 3: query working space of getrf */
    status = hipsolverDnDgetrf_bufferSize(
        cusolverH,
        n,
        n,
        d_A,
        n,
        &lwork);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    cuda_error = hipMalloc((void**)&d_work, sizeof(double)*lwork);
    assert(hipSuccess == cuda_error);

    /* step 4: LU factorization */
    status = hipsolverDnDgetrf(
        cusolverH,
        n,
        n,
        d_A,
        n,
        d_work,
        d_pivot,
        d_info);
    cuda_error = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cuda_error);

    cuda_error = hipMemcpy(Ipivot , d_pivot, sizeof(int)*n,
                            hipMemcpyDeviceToHost);
    assert(hipSuccess == cuda_error);

    cuda_error = hipMemcpy(LU, d_A, sizeof(double)*n*n,
                           hipMemcpyDeviceToHost);
    assert(hipSuccess == cuda_error);

    cuda_error = hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cuda_error);

    if ( 0 > info ){
        printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }


    /*
     * step 5: solve A*X = B
     *       | 1 |       | -0.3333 |
     *   B = | 2 |,  X = |  0.6667 |
     *       | 3 |       |  0      |
     *
     */
    status = hipsolverDnDgetrs(
        cusolverH,
        HIPBLAS_OP_N,
        n,
        1, /* nrhs */
        d_A,
        n,
        d_pivot,
        d_b,
        n,
        d_info);

    cuda_error = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cuda_error);

    cuda_error = hipMemcpy(x, d_b, sizeof(double)*n,
                            hipMemcpyDeviceToHost);
    assert(hipSuccess == cuda_error);

    /* free resources */
    if (d_A    ) hipFree(d_A);
    if (d_b    ) hipFree(d_b);
    if (d_pivot) hipFree(d_pivot);
    if (d_info ) hipFree(d_info);
    if (d_work ) hipFree(d_work);

    if(LU)
        free(LU);
    if(Ipivot)
        free(Ipivot);

    if (cusolverH   ) hipsolverDnDestroy(cusolverH);
    if (stream      ) hipStreamDestroy(stream);

    //hipDeviceReset();
}


}