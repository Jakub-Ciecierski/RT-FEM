#include "RTFEM/GPU/DeviceProperties/DevicesProperties.cuh"

#include "hip/hip_runtime.h"

namespace rtfem {

void DevicesProperties::Update(){
    int device_count;
    hipGetDeviceCount(&device_count);

    device_properties_.clear();
    for(int i = 0; i < device_count; i++){
        hipDeviceProp_t properties;
        hipGetDeviceProperties(&properties, i);

        device_properties_.push_back(DeviceProperties{properties});
    }
}

DeviceProperties* DevicesProperties::begin(){
    return &(device_properties_[0]);
}

DeviceProperties* DevicesProperties::end(){
    return &(device_properties_[device_properties_.size()]);
}

const DeviceProperties* DevicesProperties::begin() const{
    return &(device_properties_[0]);
}

const DeviceProperties* DevicesProperties::end() const{
    return &(device_properties_[device_properties_.size()]);
}

}