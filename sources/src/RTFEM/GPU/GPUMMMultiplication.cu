#include "RTFEM/GPU/GPUMMMultiplication.cuh"

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <assert.h>
#include <stdexcept>

namespace rtfem {

template <class T>
GPUMMMultiplication<T>::GPUMMMultiplication(){}

template <class T>
GPUMMMultiplication<T>::~GPUMMMultiplication(){}

template <class T>
void GPUMMMultiplication<T>::Solve(const T* A, const T* B, T* C,
                                   T alpha, T beta,
                                   int m, int k, int n,
                                   MatrixOperation A_operation,
                                   MatrixOperation B_operation){
    hipError_t cuda_error;
    hipblasStatus_t status;
    hipblasHandle_t handle;

    auto GetOperation = [](const MatrixOperation& operation){
        switch(operation){
            case MatrixOperation::None:
                return HIPBLAS_OP_N;
            case MatrixOperation::Transpose:
                return HIPBLAS_OP_T;
            default:
                return HIPBLAS_OP_N;
        }
    };

    T *d_A = nullptr;
    T *d_B = nullptr;
    T *d_C = nullptr;

    cuda_error = hipMalloc((void **) &d_A, m * k * sizeof(*A));
    assert(hipSuccess == cuda_error);
    cuda_error = hipMalloc((void **) &d_B, k * n * sizeof(*B));
    assert(hipSuccess == cuda_error);
    cuda_error = hipMalloc((void **) &d_C, m * n * sizeof(*C));
    assert(hipSuccess == cuda_error);

    status = hipblasCreate(&handle);
    assert(HIPBLAS_STATUS_SUCCESS == status);
    status = hipblasSetMatrix(m, k, sizeof(*A), A, m, d_A, m);
    assert(HIPBLAS_STATUS_SUCCESS == status);
    status = hipblasSetMatrix(k, n, sizeof(*B), B, k, d_B, k);
    assert(HIPBLAS_STATUS_SUCCESS == status);
    status = hipblasSetMatrix(m, n, sizeof(*C), C, m, d_C, m);
    assert(HIPBLAS_STATUS_SUCCESS == status);

    status = hipblasDgemm(handle,
                         GetOperation(A_operation),
                         GetOperation(B_operation),
                         m, n, k,
                         &alpha,
                         d_A, m, d_B, k,
                         &beta, d_C, m);
    assert(HIPBLAS_STATUS_SUCCESS == status);

    status = hipblasGetMatrix(m, n, sizeof(*C), d_C, m, C, m);
    assert(HIPBLAS_STATUS_SUCCESS == status);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipblasDestroy(handle);
}

template<>
void GPUMMMultiplication<float>::Solve(const float *A, const float *B, float *C,
                                       float alpha, float beta,
                                       int m, int k, int n,
                                       MatrixOperation A_operation,
                                       MatrixOperation B_operation) {
    throw std::invalid_argument(
            "GPUMMMultiplication<float>::Solve not implemented");
}

template
class GPUMMMultiplication<double>;
template
class GPUMMMultiplication<float>;

}