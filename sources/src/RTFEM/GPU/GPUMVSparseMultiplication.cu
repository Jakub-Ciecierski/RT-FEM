#include "RTFEM/GPU/GPUMVSparseMultiplication.cuh"

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <assert.h>

#include <RTFEM/DataStructure/SparseMatrixCSR.h>
#include <stdexcept>

namespace rtfem {

template<class T>
GPUMVSparseMultiplication<T>::GPUMVSparseMultiplication() : N(0),
                                                            nnz(0),
                                                            d_col(nullptr),
                                                            d_row(nullptr),
                                                            d_val(nullptr),
                                                            d_x(nullptr),
                                                            d_y(nullptr) {}

template<class T>
GPUMVSparseMultiplication<T>::~GPUMVSparseMultiplication(){
    Terminate();
}

template<class T>
void GPUMVSparseMultiplication<T>::PreSolve(const SparseMatrixCSR<T>& A){
    nnz = A.values().size();
    N = A.n();

    hipsparseStatus_t cusparseStatus;
    cusparseStatus = hipsparseCreate(&cusparseHandle);
    assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);

    cusparseStatus = hipsparseCreateMatDescr(&description);
    assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);

    switch(A.type()){
        case MatrixType::General:
            hipsparseSetMatType(description, HIPSPARSE_MATRIX_TYPE_GENERAL);
            break;
        case MatrixType::Symmetric:
            hipsparseSetMatType(description, HIPSPARSE_MATRIX_TYPE_SYMMETRIC);
            break;
    }
    hipsparseSetMatIndexBase(description, HIPSPARSE_INDEX_BASE_ZERO);

    hipError_t cuda_error;
    cuda_error = hipMalloc((void **)&d_col, nnz*sizeof(int));
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc((void **)&d_row, (N+1)*sizeof(int));
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc((void **)&d_val, nnz*sizeof(T));
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc((void **)&d_x, N*sizeof(T));
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc((void **)&d_y, N*sizeof(T));
    assert(cuda_error == hipSuccess);

    hipMemcpy(d_col, A.columns_indices().data(),
               nnz*sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(d_row, A.row_extents().data(),
               (N+1)*sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(d_val, A.values().data(),
               nnz*sizeof(T),
               hipMemcpyHostToDevice);
}

template<>
void GPUMVSparseMultiplication<float>::PreSolve(
        const SparseMatrixCSR<float>& A){
    throw std::invalid_argument(
            "GPUMVSparseMultiplication<float>::Solve not implemented");
}

template<class T>
void GPUMVSparseMultiplication<T>::Solve(
        T* x, T alpha,
        T* y, T beta){
    hipError_t cuda_error;

    cuda_error = hipMemcpy(d_x, x, N*sizeof(T), hipMemcpyHostToDevice);
    assert(cuda_error == hipSuccess);
    cuda_error = hipMemcpy(d_y, y, N*sizeof(T), hipMemcpyHostToDevice);
    assert(cuda_error == hipSuccess);

    hipsparseStatus_t cusparseStatus;
    cusparseStatus = hipsparseDcsrmv(cusparseHandle,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    N, N, nnz,
                                    &alpha,
                                    description,
                                    d_val, d_row, d_col,
                                    d_x,
                                    &beta,
                                    d_y);
    cuda_error = hipDeviceSynchronize();
    assert(cuda_error == hipSuccess);
    assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);

    cuda_error = hipMemcpy(y, d_y, sizeof(T)*N,
                            hipMemcpyDeviceToHost);
    assert(cuda_error == hipSuccess);
}

template<>
void GPUMVSparseMultiplication<float>::Solve(
        float* x, float alpha,
        float* y, float beta){
    throw std::invalid_argument(
            "GPUMVSparseMultiplication<float>::Solve not implemented");
}

template<class T>
void GPUMVSparseMultiplication<T>::Terminate(){
    hipsparseDestroy(cusparseHandle);

    hipFree(d_col);
    hipFree(d_row);
    hipFree(d_val);
    hipFree(d_x);
    hipFree(d_y);
}

template
class GPUMVSparseMultiplication<double>;
template
class GPUMVSparseMultiplication<float>;

}