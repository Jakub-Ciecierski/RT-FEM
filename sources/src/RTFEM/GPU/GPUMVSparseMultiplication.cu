#include "RTFEM/GPU/GPUMVSparseMultiplication.cuh"

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <assert.h>

#include <RTFEM/DataStructure/SparseMatrixCSR.h>

namespace rtfem {

template<class T>
void GPUMVSparseMultiplication<T>::Solve(
        const SparseMatrixCSR<T>& A,
        T* x, T alpha,
        T* y, T beta){
    int *d_col, *d_row;
    double *d_val;
    double *d_x;
    double *d_y;

    int nz = A.values().size();
    int N = A.n();

    hipblasHandle_t cublasHandle = 0;
    hipblasStatus_t hipblasStatus_t;
    hipError_t cuda_error;

    hipblasStatus_t = hipblasCreate(&cublasHandle);
    assert(hipblasStatus_t == HIPBLAS_STATUS_SUCCESS);

    hipsparseHandle_t cusparseHandle = 0;
    hipsparseStatus_t cusparseStatus;

    cusparseStatus = hipsparseCreate(&cusparseHandle);
    assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);

    hipsparseMatDescr_t descr = 0;
    cusparseStatus = hipsparseCreateMatDescr(&descr);
    assert(cusparseStatus == HIPSPARSE_STATUS_SUCCESS);

    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    cuda_error = hipMalloc((void **)&d_col, nz*sizeof(int));
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc((void **)&d_row, (N+1)*sizeof(int));
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc((void **)&d_val, nz*sizeof(double));
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc((void **)&d_x, N*sizeof(double));
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc((void **)&y, N*sizeof(double));
    assert(cuda_error == hipSuccess);

    hipMemcpy(d_col, A.columns_indices().data(),
               nz*sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(d_row, A.row_extents().data(),
               (N+1)*sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(d_val, A.values().data(),
               nz*sizeof(double),
               hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N*sizeof(double), hipMemcpyHostToDevice);

    hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                   N, N, nz,
                   &alpha,
                   descr,
                   d_val, d_row, d_col,
                   d_x,
                   &beta, d_y);

    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);

    hipFree(d_col);
    hipFree(d_row);
    hipFree(d_val);
    hipFree(d_x);
    hipFree(d_y);
}

}